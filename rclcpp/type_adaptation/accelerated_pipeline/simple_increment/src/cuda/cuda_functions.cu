// Copyright (c) 2021, NVIDIA CORPORATION.  All rights reserved.
// Copyright 2021 Open Source Robotics Foundation, Inc.
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.


#include <hip/hip_runtime.h>
#include <cstdint>

__global__
void myinc(int size, const uint8_t * source, uint8_t * destination)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < size) {
    destination[index] = source[index] + 1;
  }
}

void cuda_compute_inc(int size, const uint8_t * source, uint8_t * destination, const hipStream_t & stream)
{
  myinc<<<64, 64, 0, stream>>>(size, source, destination);
}

void cuda_compute_inc_inplace(int size, uint8_t * image, const hipStream_t & stream)
{
  myinc<<<64, 64, 0, stream>>>(size, image, image);
}
