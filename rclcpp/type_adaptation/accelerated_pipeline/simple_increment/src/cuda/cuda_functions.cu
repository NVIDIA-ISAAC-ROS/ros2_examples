
#include <hip/hip_runtime.h>
#include <cstdint>

__global__
void myxor(int n, uint8_t *image)
{
  for (int i = 0; i < n; ++i) {
    image[i] = image[i] + 50;
  }
}

void cuda_compute_xor(int n, uint8_t * image, const hipStream_t & stream)
{
  myxor<<<1, 1, 0, stream>>>(n, image);
}

__global__
void myinc(int size, const uint8_t * source, uint8_t * destination)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < size) {
    destination[index] = source[index] + 1;
  }
}

void cuda_compute_inc(int size, const uint8_t * source, uint8_t * destination, const hipStream_t & stream)
{
  myinc<<<64, 64, 0, stream>>>(size, source, destination);
}

void cuda_compute_inc_inplace(int size, uint8_t * image, const hipStream_t & stream)
{
  myinc<<<64, 64, 0, stream>>>(size, image, image);
}
