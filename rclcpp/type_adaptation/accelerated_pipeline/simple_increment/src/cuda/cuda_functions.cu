
#include <hip/hip_runtime.h>
#include <cstdint>

__global__
void myinc(int size, const uint8_t * source, uint8_t * destination)
{
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if (index < size) {
    destination[index] = source[index] + 1;
  }
}

void cuda_compute_inc(int size, const uint8_t * source, uint8_t * destination, const hipStream_t & stream)
{
  myinc<<<64, 64, 0, stream>>>(size, source, destination);
}

void cuda_compute_inc_inplace(int size, uint8_t * image, const hipStream_t & stream)
{
  myinc<<<64, 64, 0, stream>>>(size, image, image);
}
